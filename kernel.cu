#include "hip/hip_runtime.h"
﻿// 学习运行时参数的设置：分配1个线程块，3个线程执行核函数，打印3次“hello world from GPU”

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void fun(void)
{
    printf("hello world from GPU\n");
}

int main()
{
    printf("hello world\n");

    fun << <1, 3 >> > ();
    hipDeviceSynchronize();
}
